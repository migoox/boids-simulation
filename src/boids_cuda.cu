#include "hip/hip_runtime.h"
#include "boids_cuda.hpp"
#include "hip/hip_runtime.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

using namespace boids::cuda;
using namespace boids;

__device__ hiprandState state[SimulationParameters::MAX_BOID_COUNT];

// Stores starting index of all elements in the queried cell.
__device__ int cell_start[SimulationParameters::MAX_CELL_COUNT];
__device__ int cell_end[SimulationParameters::MAX_CELL_COUNT];

__device__ CellId flatten_coords(const SimulationParameters *sim_params, CellCoords coords) {
    CellCoord grid_size_x = std::ceil(sim_params->aquarium_size.x / sim_params->distance);
    CellCoord grid_size_y = std::ceil(sim_params->aquarium_size.y / sim_params->distance);

    return coords.x + coords.y * grid_size_x + coords.z * grid_size_x * grid_size_y;
}

__device__ CellId flatten_coords(const SimulationParameters *sim_params, CellCoord x, CellCoord y, CellCoord z) {
    CellCoord grid_size_x = std::ceil(sim_params->aquarium_size.x / sim_params->distance);
    CellCoord grid_size_y = std::ceil(sim_params->aquarium_size.y / sim_params->distance);

    return x + y * grid_size_x + z * grid_size_x * grid_size_y;
}

__device__ CellCoords get_cell_cords(const SimulationParameters *sim_params, const glm::vec4& position) {
    return CellCoords {
            static_cast<CellCoord>((position.x + sim_params->aquarium_size.x / 2.f) / sim_params->distance),
            static_cast<CellCoord>((position.y + sim_params->aquarium_size.y / 2.f) / sim_params->distance),
            static_cast<CellCoord>((position.z + sim_params->aquarium_size.z / 2.f) / sim_params->distance)
    };
}

__device__ CellId get_flat_cell_id(const SimulationParameters *sim_params, const glm::vec4& position) {
    return flatten_coords(
            sim_params,
            get_cell_cords(sim_params, position)
    );
}

__global__ void setup_curand(size_t max_boid_count) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= max_boid_count) {
        return;
    }

    hiprand_init(1234, id, 0, &state[id]);
}
void check_cuda_error(const hipError_t &cuda_status, const char *msg) {
    if (cuda_status != hipSuccess) {
        std::cerr << msg << hipGetErrorString(cuda_status) << std::endl;
        std::terminate();
    }
};

GPUBoids::GPUBoids(const boids::Boids& boids, const boids::BoidsRenderer& renderer) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("[CUDA] Device %d: Compute Capability %d.%d\n", i, prop.major, prop.minor);
    }

    // Allocate memory on the device using hipMalloc
    size_t array_size_vec3 = SimulationParameters::MAX_BOID_COUNT * sizeof(glm::vec3);
    size_t array_size_vec4 = SimulationParameters::MAX_BOID_COUNT * sizeof(glm::vec4);

    // Malloc and send boids data
    hipError_t cuda_status;
    cuda_status = hipMalloc((void**)&m_dev_position, array_size_vec4);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_velocity, array_size_vec3);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_acceleration, array_size_vec3);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_orient, sizeof(BoidsOrientation));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    cuda_status = hipMemcpy(m_dev_position, boids.position, array_size_vec4, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_velocity, boids.velocity, array_size_vec3, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_acceleration, boids.acceleration, array_size_vec3, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_orient, &boids.orientation, sizeof(BoidsOrientation), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    // Prepare simulation params container
    cuda_status = hipMalloc((void**)&m_dev_sim_params, sizeof(SimulationParameters));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed ");

    // Prepare boid_id and cell_id
    cuda_status = hipMalloc((void**)&m_dev_cell_id, SimulationParameters::MAX_BOID_COUNT * sizeof(CellId));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_boid_id, SimulationParameters::MAX_BOID_COUNT * sizeof(BoidId));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    // Setup hiprand
    setup_curand<<<1024,SimulationParameters::MAX_BOID_COUNT / 1024 + 1>>>(SimulationParameters::MAX_BOID_COUNT);


}

GPUBoids::~GPUBoids() {
    hipFree(m_dev_position);
    hipFree(m_dev_velocity);
    hipFree(m_dev_orient);
    hipFree(m_dev_sim_params);
    hipFree(m_dev_cell_id);
    hipFree(m_dev_boid_id);
}

__global__ void find_cell_ids(const boids::SimulationParameters *params, BoidId *boid_id, CellId *cell_id, glm::vec4 *position, size_t boids_count) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= boids_count) return;

    boid_id[b_id] = b_id;
    cell_id[b_id] = get_flat_cell_id(params, position[b_id]);
}

__global__ void find_starts(BoidId *boid_id, CellId *cell_id, size_t boids_count) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k >= boids_count)  {
        return;
    }

    // TODO: do it better...
    if (k == 0) {
        cell_start[cell_id[0]] = 0;
    }

    if (k < boids_count - 1) {
       if (cell_id[k] != cell_id[k + 1]) {
           cell_start[cell_id[k + 1]] = k + 1;
           cell_end[cell_id[k]] = k;
       }
    } else {
        if (k == boids_count - 1) {
            cell_end[cell_id[k]] = k;
        }
    }
}

__global__ void update_acceleration(const SimulationParameters *params, BoidId *boid_id, glm::vec4 *position, glm::vec3 *velocity, glm::vec3 *acceleration, size_t boids_count) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= boids_count) return;

    glm::vec3 separation(0.);
    glm::vec3 avg_vel(0.);
    glm::vec3 avg_pos(0.);
    uint32_t neighbors_count = 0;

    CellCoords cell_coords = get_cell_cords(params, position[b_id]);

    CellCoord grid_size_x = std::ceil(params->aquarium_size.x / params->distance);
    CellCoord grid_size_y = std::ceil(params->aquarium_size.y / params->distance);
    CellCoord grid_size_z = std::ceil(params->aquarium_size.y / params->distance);

    auto z_start = static_cast<CellCoord>(max(int(cell_coords.z) - 1, 0));
    auto z_end = static_cast<CellCoord>(min(cell_coords.z + 1, grid_size_z - 1));

    auto y_start = static_cast<CellCoord>(max(int(cell_coords.y) - 1, 0));
    auto y_end = static_cast<CellCoord>(min(cell_coords.y + 1, grid_size_y - 1));

    auto x_start = static_cast<CellCoord>(max(int(cell_coords.x) - 1, 0));
    auto x_end = static_cast<CellCoord>(min(cell_coords.x + 1, grid_size_x - 1));

    for (CellCoord curr_cell_z = z_start; curr_cell_z <= z_end; ++curr_cell_z) {
        for (CellCoord curr_cell_y = y_start; curr_cell_y <= y_end; ++curr_cell_y) {
            for (CellCoord curr_cell_x = x_start; curr_cell_x <= x_end; ++curr_cell_x) {
                CellId curr_flat_id = flatten_coords(
                        params,
                        curr_cell_x,
                        curr_cell_y,
                        curr_cell_z
                );

                for (int k = cell_start[curr_flat_id]; k <= cell_end[curr_flat_id]; ++k) {
                    BoidId other_id = boid_id[k];
                    if (other_id == b_id) {
                        continue;
                    }

                    auto distance2 = glm::dot(position[b_id] - position[other_id], position[b_id] - position[other_id]);
                    if (distance2 > params->distance * params->distance) {
                        continue;
                    }

                    separation += glm::vec3(glm::normalize(position[b_id] - position[other_id]) / distance2);
                    avg_vel += velocity[other_id];
                    avg_pos += glm::vec3(position[other_id]);

                    ++neighbors_count;
                }
            }
        }
    }

    // Naive
//    for (BoidId other_id = 0; other_id < boids_count; ++other_id) {
//        if (other_id == b_id) {
//            continue;
//        }
//
//        auto distance2 = glm::dot(position[b_id] - position[other_id], position[b_id] - position[other_id]);
//        if (distance2 > params->distance * params->distance) {
//            continue;
//        }
//
////        separation += glm::normalize(position[b_id] - position[other_id]) / distance2;
////        avg_vel += velocity[other_id];
////        avg_pos += position[other_id];
//
//        ++neighbors_count;
//    }

    if (neighbors_count > 0) {
        avg_vel /= float(neighbors_count);
        avg_pos /= float(neighbors_count);
    }

     //printf("%d vs %d\n", neighbors_count, neighbors_count2);
//    for (int i = 0; i < neighbors_count; i++) {
//        printf("%u SORT %u\n", b_id, test[i]);
//    }
//    for (int i = 0; i < neighbors_count2; i++) {
//        printf("%u NAIVE %u\n", b_id, test2[i]);
//    }

    // Final acceleration of the current boid
    acceleration[b_id] =
            params->separation * separation +
            params->alignment * (avg_vel - velocity[b_id]) +
            params->cohesion * (avg_pos - glm::vec3(position[b_id]));

    // Add noise
    hiprandState local_state = state[b_id];
    float x = hiprand_uniform(&local_state);
    float y = hiprand_uniform(&local_state);
    float z = hiprand_uniform(&local_state);
    state[b_id] = local_state;
    acceleration[b_id] += glm::normalize(glm::vec3(2.f * (x - 0.5f), 2.f * (y - 0.5f), 2.f * (z - 0.5f))) * params->noise;
}

__global__ void update_simulation(const SimulationParameters *params, glm::vec4 *position, glm::vec3 *velocity, glm::vec3 *acceleration, BoidsOrientation *orient, size_t boids_count, float dt) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= boids_count) return;

    // TODO: Parametrize wall and wall_force values
    float wall = 4.f;
    float wall_acc = 15.f;
    if (position[b_id].x > params->aquarium_size.x / 2.f - wall) {
        auto intensity = std::abs((params->aquarium_size.x / 2.f - wall - position[b_id].x) / wall);
        acceleration[b_id] += intensity * glm::vec3(-wall_acc, 0.f, 0.f);
    } else if (position[b_id].x < -params->aquarium_size.x / 2.f + wall) {
        auto intensity = std::abs((-params->aquarium_size.x / 2.f + wall - position[b_id].x) / wall);
        acceleration[b_id] += intensity * glm::vec3(wall_acc, 0.f, 0.f);
    }

    if (position[b_id].y > params->aquarium_size.y / 2.f - wall) {
        auto intensity = std::abs((params->aquarium_size.y / 2.f - wall - position[b_id].y) / wall);
        acceleration[b_id] += intensity * glm::vec3(0.f, -wall_acc, 0.f);
    } else if (position[b_id].y < -params->aquarium_size.y / 2.f + wall) {
        auto intensity = std::abs((-params->aquarium_size.y / 2.f + wall - position[b_id].y) / wall);
        acceleration[b_id] += intensity * glm::vec3(0.f, wall_acc, 0.f);
    }

    if (position[b_id].z > params->aquarium_size.z / 2.f - wall) {
        auto intensity = std::abs((params->aquarium_size.z / 2.f - wall - position[b_id].z) / wall);
        acceleration[b_id] += intensity * glm::vec3(0.f, 0.f, -wall_acc);
    } else if (position[b_id].z < -params->aquarium_size.z / 2.f + wall) {
        auto intensity = std::abs((-params->aquarium_size.z / 2.f + wall - position[b_id].z) / wall);
        acceleration[b_id] += intensity * glm::vec3(0.f, 0.f, wall_acc);
    }

    velocity[b_id] += acceleration[b_id] * dt;

    if (glm::length(velocity[b_id]) > params->max_speed) {
        velocity[b_id] = glm::normalize(velocity[b_id]) * params->max_speed;
    } else if (glm::length(velocity[b_id]) < params->min_speed){
        velocity[b_id] = glm::normalize(velocity[b_id]) * params->min_speed;
    }

    position[b_id] += glm::vec4(velocity[b_id] * dt, 0.f);

    // Update orientation
    orient->forward[b_id] = glm::vec4(glm::normalize(velocity[b_id]), 0.f);
    orient->right[b_id] = glm::vec4(glm::normalize(
            glm::cross(glm::vec3(orient->up[b_id]),glm::vec3(orient->forward[b_id]))
    ), 0.f);

    orient->up[b_id] = glm::vec4(glm::normalize(
            glm::cross(glm::vec3(orient->forward[b_id]),glm::vec3(orient->right[b_id]))
    ), 0.f);
}

void GPUBoids::update_simulation_with_sort(const boids::SimulationParameters &params, Boids &boids, float dt) {
    // 1. Update simulation parameters
    hipError_t cuda_status = hipMemcpy(m_dev_sim_params, &params, sizeof(boids::SimulationParameters), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    size_t threads_per_block = 1024;
    size_t blocks_num = SimulationParameters::MAX_BOID_COUNT / threads_per_block + 1; // TODO

    // 1.
    find_cell_ids<<<blocks_num, threads_per_block>>>(m_dev_sim_params, m_dev_boid_id, m_dev_cell_id, m_dev_position, SimulationParameters::MAX_BOID_COUNT);
    hipDeviceSynchronize();

    // 2.
    thrust::sort_by_key(thrust::device, m_dev_cell_id, m_dev_cell_id + SimulationParameters::MAX_BOID_COUNT, m_dev_boid_id);
    hipDeviceSynchronize();

    // 3.
    find_starts<<<blocks_num, threads_per_block>>>(m_dev_boid_id, m_dev_cell_id, SimulationParameters::MAX_BOID_COUNT);
    hipDeviceSynchronize();

    // 4.
    update_acceleration<<<blocks_num, threads_per_block>>>(m_dev_sim_params, m_dev_boid_id, m_dev_position, m_dev_velocity, m_dev_acceleration, SimulationParameters::MAX_BOID_COUNT);
    hipDeviceSynchronize();

    // 5.
    update_simulation<<<blocks_num, threads_per_block>>>(m_dev_sim_params, m_dev_position, m_dev_velocity, m_dev_acceleration, m_dev_orient, SimulationParameters::MAX_BOID_COUNT, dt);
    hipDeviceSynchronize();

    // TODO: use direct CUDA -> OPENGL
    int arr_size = sizeof(glm::vec3) * SimulationParameters::MAX_BOID_COUNT;
    cuda_status = hipMemcpy(boids.position, m_dev_position, arr_size, hipMemcpyDeviceToHost);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(&boids.orientation, m_dev_orient, arr_size, hipMemcpyDeviceToHost);
}
