#include "hip/hip_runtime.h"
#include "boids_cuda.hpp"
#include <iostream>

__device__ boids::cuda::CellId
boids::cuda::flatten_coords(const boids::SimulationParameters &sim_params, boids::cuda::CellCoord x,
                                          boids::cuda::CellCoord y, boids::cuda::CellCoord z) {
    float cell_size = 2 * sim_params.distance;
    CellCoord grid_size_x = std::ceil(sim_params.aquarium_size.x / cell_size);
    CellCoord grid_size_y = std::ceil(sim_params.aquarium_size.y / cell_size);

    return x + y * grid_size_x + z * grid_size_x * grid_size_y;
}

boids::cuda::GPUBoids::GPUBoids(const boids::Boids& boids) {
    // Allocate memory on the device using hipMalloc
//    hipError_t cudaStatus = hipMalloc((void**)&deviceMemory, size);
//
//    if (cudaStatus != hipSuccess) {
//        std::cerr << "[CUDA]: hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
//        return 1;
//    }
}
