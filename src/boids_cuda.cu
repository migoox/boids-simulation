#include "hip/hip_runtime.h"
#include "boids_cuda.hpp"
#include "hip/hip_runtime.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>

using namespace boids::cuda;
using namespace boids;

__device__ hiprandState state[SimulationParameters::MAX_BOID_COUNT];

__device__ CellId flatten_coords(const SimulationParameters *sim_params, CellCoords coords) {
    CellCoord grid_size_x = std::ceil(sim_params->aquarium_size.x / sim_params->distance);
    CellCoord grid_size_y = std::ceil(sim_params->aquarium_size.y / sim_params->distance);

    return coords.x + coords.y * grid_size_x + coords.z * grid_size_x * grid_size_y;
}

__device__ CellId flatten_coords(const SimulationParameters *sim_params, CellCoord x, CellCoord y, CellCoord z) {
    CellCoord grid_size_x = std::ceil(sim_params->aquarium_size.x / sim_params->distance);
    CellCoord grid_size_y = std::ceil(sim_params->aquarium_size.y / sim_params->distance);

    return x + y * grid_size_x + z * grid_size_x * grid_size_y;
}

__device__ CellCoords get_cell_cords(const SimulationParameters *sim_params, const glm::vec4& position) {
    return CellCoords {
            static_cast<CellCoord>((position.x + sim_params->aquarium_size.x / 2.f) / sim_params->distance),
            static_cast<CellCoord>((position.y + sim_params->aquarium_size.y / 2.f) / sim_params->distance),
            static_cast<CellCoord>((position.z + sim_params->aquarium_size.z / 2.f) / sim_params->distance)
    };
}

__device__ CellId get_flat_cell_id(const SimulationParameters *sim_params, const glm::vec4& position) {
    return flatten_coords(
            sim_params,
            get_cell_cords(sim_params, position)
    );
}

__device__ void update_orientation(BoidsOrientation *orient, glm::vec3 *velocity, BoidId b_id) {
    if (glm::dot(glm::vec3(orient->up[b_id]), glm::vec3(0.f, 1.f, 0.f)) < 0.f) {
        orient->up[b_id] = glm::vec4(0.f, 1.f, 0.f, 0.f);
    }

    // Update orientation
    orient->forward[b_id] = glm::vec4(glm::normalize(velocity[b_id]), 0.f);
    orient->right[b_id] = glm::vec4(glm::normalize(
            glm::cross(glm::vec3(orient->up[b_id]),glm::vec3(orient->forward[b_id]))
    ), 0.f);

    orient->up[b_id] = glm::vec4(glm::normalize(
            glm::cross(glm::vec3(orient->forward[b_id]),glm::vec3(orient->right[b_id]))
    ), 0.f);

}

__device__ void update_pos_vel(
        const SimulationParameters *params,
        const BoidId b_id,
        glm::vec4 *position,
        glm::vec4 *position_old,
        glm::vec3 *velocity,
        glm::vec3 *velocity_old,
        glm::vec3 acceleration,
        BoidsOrientation *orient,
        float dt
) {
    float wall = 4.f;
    float wall_acc = 15.f;

    if (position_old[b_id].x > params->aquarium_size.x / 2.f - wall) {
        auto intensity = std::abs((params->aquarium_size.x / 2.f - wall - position_old[b_id].x) / wall);
        acceleration += intensity * glm::vec3(-wall_acc, 0.f, 0.f);
    } else if (position_old[b_id].x < -params->aquarium_size.x / 2.f + wall) {
        auto intensity = std::abs((-params->aquarium_size.x / 2.f + wall - position_old[b_id].x) / wall);
        acceleration += intensity * glm::vec3(wall_acc, 0.f, 0.f);
    }

    if (position_old[b_id].y > params->aquarium_size.y / 2.f - wall) {
        auto intensity = std::abs((params->aquarium_size.y / 2.f - wall - position_old[b_id].y) / wall);
        acceleration += intensity * glm::vec3(0.f, -wall_acc, 0.f);
    } else if (position_old[b_id].y < -params->aquarium_size.y / 2.f + wall) {
        auto intensity = std::abs((-params->aquarium_size.y / 2.f + wall - position_old[b_id].y) / wall);
        acceleration += intensity * glm::vec3(0.f, wall_acc, 0.f);
    }

    if (position_old[b_id].z > params->aquarium_size.z / 2.f - wall) {
        auto intensity = std::abs((params->aquarium_size.z / 2.f - wall - position_old[b_id].z) / wall);
        acceleration += intensity * glm::vec3(0.f, 0.f, -wall_acc);
    } else if (position_old[b_id].z < -params->aquarium_size.z / 2.f + wall) {
        auto intensity = std::abs((-params->aquarium_size.z / 2.f + wall - position_old[b_id].z) / wall);
        acceleration += intensity * glm::vec3(0.f, 0.f, wall_acc);
    }

    velocity[b_id] = velocity_old[b_id] + acceleration * dt;

    if (glm::length(velocity[b_id]) > params->max_speed) {
        velocity[b_id] = glm::normalize(velocity[b_id]) * params->max_speed;
    } else if (glm::length(velocity[b_id]) < params->min_speed){
        velocity[b_id] = glm::normalize(velocity[b_id]) * params->min_speed;
    }

    position[b_id] = position_old[b_id] + glm::vec4(velocity[b_id] * dt, 0.f);

    // Update orientation
    update_orientation(orient, velocity, b_id);
}

__global__ void setup_curand(size_t max_boid_count) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= max_boid_count) {
        return;
    }

    hiprand_init(1234, id, 0, &state[id]);
}


__global__ void ker_find_cell_ids(const boids::SimulationParameters *params, BoidId *boid_id, CellId *cell_id, glm::vec4 *position_old) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= params->boids_count) return;

    boid_id[b_id] = b_id;
    cell_id[b_id] = get_flat_cell_id(params, position_old[b_id]);
}

__global__ void ker_find_starts(CellId *cell_id, int *cell_start, int *cell_end, size_t boids_count) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k >= boids_count)  {
        return;
    }

    // TODO: do it better...
    if (k == 0) {
        cell_start[cell_id[0]] = 0;
    }

    if (k < boids_count - 1) {
        if (cell_id[k] != cell_id[k + 1]) {
            cell_start[cell_id[k + 1]] = k + 1;
            cell_end[cell_id[k]] = k + 1;
        }
    } else {
        if (k == boids_count - 1) {
            cell_end[cell_id[k]] = k + 1;
        }
    }
}


__global__ void ker_clear_starts(CellId *cell_id, int *cell_start, int *cell_end, size_t boids_count) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= boids_count) {
        return;
    }

    cell_start[cell_id[k]] = 0;
    cell_end[cell_id[k]] = 0;
}



__global__ void ker_update_simulation_naive(
        const SimulationParameters *params,
        glm::vec4 *position,
        glm::vec4 *position_old,
        glm::vec3 *velocity,
        glm::vec3 *velocity_old,
        BoidsOrientation *orient,
        float dt
) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= params->boids_count) return;

    glm::vec3 acceleration(0.f);

    glm::vec3 separation(0.);
    glm::vec3 avg_vel(0.);
    glm::vec3 avg_pos(0.);
    uint32_t neighbors_count = 0;

    for (BoidId other_id = 0; other_id < params->boids_count; ++other_id) {
        if (other_id == b_id) {
            continue;
        }

        auto distance2 = glm::dot(position_old[b_id] - position_old[other_id], position_old[b_id] - position_old[other_id]);
        if (distance2 > params->distance * params->distance) {
            continue;
        }

        separation += glm::vec3(glm::normalize(position_old[b_id] - position_old[other_id]) / distance2);
        avg_vel += velocity_old[other_id];
        avg_pos += glm::vec3(position_old[other_id]);

        ++neighbors_count;
    }

    if (neighbors_count > 0) {
        avg_vel /= float(neighbors_count);
        avg_pos /= float(neighbors_count);

        // Final acceleration of the current boid
        acceleration =
                params->separation * separation +
                params->alignment * (avg_vel - velocity_old[b_id]) +
                params->cohesion * (avg_pos - glm::vec3(position_old[b_id]));
    }

    // Add noise
    hiprandState local_state = state[b_id];
    float x = hiprand_uniform(&local_state);
    float y = hiprand_uniform(&local_state);
    float z = hiprand_uniform(&local_state);
    state[b_id] = local_state;
    acceleration += glm::normalize(glm::vec3(2.f * (x - 0.5f), 2.f * (y - 0.5f), 2.f * (z - 0.5f))) * params->noise;

    // Update pos and vel
    update_pos_vel(
            params,
            b_id,
            position,
            position_old,
            velocity,
            velocity_old,
            acceleration,
            orient,
            dt
    );
}

__global__ void ker_update_simulation_with_sort(
        const SimulationParameters *params,
        const BoidId *boid_id,
        const int *cell_start,
        const int *cell_end,
        glm::vec4 *position,
        glm::vec4 *position_old,
        glm::vec3 *velocity,
        glm::vec3 *velocity_old,
        BoidsOrientation *orient,
        float dt
) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= params->boids_count) return;

    glm::vec3 acceleration(0.f);

    glm::vec3 separation(0.);
    glm::vec3 avg_vel(0.);
    glm::vec3 avg_pos(0.);
    uint32_t neighbors_count = 0;

    CellCoords cell_coords = get_cell_cords(params, position_old[b_id]);

    CellCoord grid_size_x = std::ceil(params->aquarium_size.x / params->distance);
    CellCoord grid_size_y = std::ceil(params->aquarium_size.y / params->distance);
    CellCoord grid_size_z = std::ceil(params->aquarium_size.z / params->distance);

    auto x_start = static_cast<CellCoord>(max(int(cell_coords.x) - 1, 0));
    auto x_end = static_cast<CellCoord>(min(cell_coords.x + 1, grid_size_x - 1));

    auto y_start = static_cast<CellCoord>(max(int(cell_coords.y) - 1, 0));
    auto y_end = static_cast<CellCoord>(min(cell_coords.y + 1, grid_size_y - 1));

    auto z_start = static_cast<CellCoord>(max(int(cell_coords.z) - 1, 0));
    auto z_end = static_cast<CellCoord>(min(cell_coords.z + 1, grid_size_z - 1));

    for (CellCoord curr_cell_z = z_start; curr_cell_z <= z_end; ++curr_cell_z) {
        for (CellCoord curr_cell_y = y_start; curr_cell_y <= y_end; ++curr_cell_y) {
            for (CellCoord curr_cell_x = x_start; curr_cell_x <= x_end; ++curr_cell_x) {
                CellId curr_flat_id = flatten_coords(
                        params,
                        curr_cell_x,
                        curr_cell_y,
                        curr_cell_z
                );

                for (int k = cell_start[curr_flat_id]; k < cell_end[curr_flat_id]; ++k) {
                    BoidId other_id = boid_id[k];

                    if (other_id == b_id) {
                        continue;
                    }

                    auto distance2 = glm::dot(position_old[b_id] - position_old[other_id], position_old[b_id] - position_old[other_id]);
                    if (distance2 > params->distance * params->distance) {
                        continue;
                    }

                    separation += glm::vec3(glm::normalize(position_old[b_id] - position_old[other_id]) / distance2);
                    avg_vel += velocity_old[other_id];
                    avg_pos += glm::vec3(position_old[other_id]);

                    ++neighbors_count;
                }
            }
        }
    }

    if (neighbors_count > 0) {
        avg_vel /= float(neighbors_count);
        avg_pos /= float(neighbors_count);

        // Final acceleration of the current boid
        acceleration =
                params->separation * separation +
                params->alignment * (avg_vel - velocity_old[b_id]) +
                params->cohesion * (avg_pos - glm::vec3(position_old[b_id]));
    }

    // Add noise
    hiprandState local_state = state[b_id];
    float x = hiprand_uniform(&local_state);
    float y = hiprand_uniform(&local_state);
    float z = hiprand_uniform(&local_state);
    state[b_id] = local_state;
    acceleration += glm::normalize(glm::vec3(2.f * (x - 0.5f), 2.f * (y - 0.5f), 2.f * (z - 0.5f))) * params->noise;

    // Update pos and vel
    update_pos_vel(
            params,
            b_id,
            position,
            position_old,
            velocity,
            velocity_old,
            acceleration,
            orient,
            dt
    );
}

__global__ void ker_reset_simulation(
        const SimulationParameters *params,
        glm::vec4 *position_old,
        glm::vec3 *velocity_old,
        BoidsOrientation *orient
) {
    BoidId b_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (b_id >= params->boids_count) return;

    hiprandState local_state = state[b_id];

    float x = (hiprand_uniform(&local_state) - 0.5f) * params->aquarium_size.x;
    float y = (hiprand_uniform(&local_state) - 0.5f) * params->aquarium_size.y;
    float z = (hiprand_uniform(&local_state) - 0.5f) * params->aquarium_size.z;

    position_old[b_id] = glm::vec4(x, y, z, 1.f);

    orient->forward[b_id] = glm::vec4(0.f, 0.f, 1.f, 0.f);
    orient->up[b_id] = glm::vec4(0.f, 1.f, 0.f, 0.f);
    orient->right[b_id] = glm::vec4(1.f, 0.f, 0.f, 0.f);

    x = (hiprand_uniform(&local_state) - 0.5f) * 2.f;
    y = (hiprand_uniform(&local_state) - 0.5f) * 2.f;
    z = (hiprand_uniform(&local_state) - 0.5f) * 2.f;

    velocity_old[b_id] = glm::vec3(0.05f * glm::normalize(glm::vec3(x, y, z)));

    // Update basis vectors (orientation)
    update_orientation(orient, velocity_old, b_id);

    state[b_id] = local_state;
}

__global__ void init_starts(int *cell_start, int *cell_end, size_t count) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= count) {
        return;
    }
    cell_start[k] = 0;
    cell_end[k] = 0;
}

void check_cuda_error(const hipError_t &cuda_status, const char *msg) {
    if (cuda_status != hipSuccess) {
        std::cerr << msg << hipGetErrorString(cuda_status) << std::endl;
        std::terminate();
    }
}

GPUBoids::GPUBoids(const boids::Boids& boids, const boids::BoidsRenderer& renderer) {
    hipError_t cuda_err;
    int gl_device_id;
    unsigned int gl_device_count;
    // Try to find and set opengl device
    cuda_err = hipGLGetDevices(&gl_device_count,&gl_device_id,1,hipGLDeviceListAll);
    cuda_err = hipSetDevice(gl_device_id);
    if (cuda_err == hipSuccess) {
        std::cout << "[CUDA] Found cuda device attached to the current OpenGL context: " << gl_device_id << ". GL buffers registration are not currently supported.\n";
        // TODO: register gl buffers
        this->init_default(boids);
    } else {
        std::cout << "[CUDA]: Couldn't find any cuda device attached to teh current OpenGL context, GL buffers aren't going to be registered.\n";
        this->init_default(boids);
    }
}

GPUBoids::GPUBoids(const Boids &boids) {
    this->init_default(boids);
}

void GPUBoids::init_default(const Boids& boids) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("[CUDA] Device %d: Compute Capability %d.%d\n", i, prop.major, prop.minor);
    }

    // Allocate memory on the device using hipMalloc
    size_t array_size_vec3 = SimulationParameters::MAX_BOID_COUNT * sizeof(glm::vec3);
    size_t array_size_vec4 = SimulationParameters::MAX_BOID_COUNT * sizeof(glm::vec4);

    // Malloc and send boids data
    hipError_t cuda_status;
    cuda_status = hipMalloc((void**)&m_dev_position, array_size_vec4);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_velocity, array_size_vec3);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_position_old, array_size_vec4);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_velocity_old, array_size_vec3);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_orient, sizeof(BoidsOrientation));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    cuda_status = hipMemcpy(m_dev_position_old, boids.position, array_size_vec4, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_velocity_old, boids.velocity, array_size_vec3, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_orient, &boids.orientation, sizeof(BoidsOrientation), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    // Prepare simulation params container
    cuda_status = hipMalloc((void**)&m_dev_sim_params, sizeof(SimulationParameters));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed ");

    // Prepare boid_id and cell_id
    cuda_status = hipMalloc((void**)&m_dev_cell_id, SimulationParameters::MAX_BOID_COUNT * sizeof(CellId));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_boid_id, SimulationParameters::MAX_BOID_COUNT * sizeof(BoidId));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    // Prepare start and end arrays
    cuda_status = hipMalloc((void**)&m_dev_cell_start, SimulationParameters::MAX_CELL_COUNT * sizeof(int));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_cell_end, SimulationParameters::MAX_CELL_COUNT * sizeof(int));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    init_starts<<<1024, SimulationParameters::MAX_CELL_COUNT / 1024 + 1>>>(m_dev_cell_start, m_dev_cell_end, SimulationParameters::MAX_CELL_COUNT);

    // Setup hiprand
    setup_curand<<<1024,SimulationParameters::MAX_BOID_COUNT / 1024 + 1>>>(SimulationParameters::MAX_BOID_COUNT);
}

void GPUBoids::init_with_gl(const Boids &boids, const BoidsRenderer &renderer) {
    // TODO
    std::terminate();
}

GPUBoids::~GPUBoids() {
    hipFree(m_dev_position);
    hipFree(m_dev_velocity);
    hipFree(m_dev_orient);
    hipFree(m_dev_sim_params);
    hipFree(m_dev_cell_id);
    hipFree(m_dev_boid_id);
}

void GPUBoids::update_simulation_naive(const boids::SimulationParameters &params, Boids &boids, float dt) {
    // 1. Update simulation parameters
    hipError_t cuda_status = hipMemcpy(m_dev_sim_params, &params, sizeof(boids::SimulationParameters), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    size_t threads_per_block = 1024;
    size_t blocks_num = params.boids_count / threads_per_block + 1;

    ker_update_simulation_naive<<<blocks_num, threads_per_block>>>(
            m_dev_sim_params,
            m_dev_position,
            m_dev_position_old,
            m_dev_velocity,
            m_dev_velocity_old,
            m_dev_orient,
            dt
    );
    hipDeviceSynchronize();

    move_boids_data_to_cpu(boids);
    swap_buffers();
}

void GPUBoids::update_simulation_with_sort(const boids::SimulationParameters &params, Boids &boids, float dt) {
    // 1. Update simulation parameters
    hipError_t cuda_status = hipMemcpy(m_dev_sim_params, &params, sizeof(boids::SimulationParameters), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    size_t threads_per_block = 1024;
    size_t blocks_num = params.boids_count / threads_per_block + 1;

    // 1.
    ker_find_cell_ids<<<blocks_num, threads_per_block>>>(
            m_dev_sim_params,
            m_dev_boid_id,
            m_dev_cell_id,
            m_dev_position_old
    );
    hipDeviceSynchronize();

    // 2.
    thrust::sort_by_key(
            thrust::device,
            m_dev_cell_id,
            m_dev_cell_id + params.boids_count,
            m_dev_boid_id
    );
    hipDeviceSynchronize();

    // 3.
    ker_find_starts<<<blocks_num, threads_per_block>>>(
            m_dev_cell_id,
            m_dev_cell_start,
            m_dev_cell_end,
            params.boids_count
    );
    hipDeviceSynchronize();

    // 4.
    ker_update_simulation_with_sort<<<blocks_num, threads_per_block>>>(
            m_dev_sim_params,
            m_dev_boid_id,
            m_dev_cell_start,
            m_dev_cell_end,
            m_dev_position,
            m_dev_position_old,
            m_dev_velocity,
            m_dev_velocity_old,
            m_dev_orient,
            dt
    );
    hipDeviceSynchronize();

    // 5.
    ker_clear_starts<<<blocks_num, threads_per_block>>>(
            m_dev_cell_id,
            m_dev_cell_start,
            m_dev_cell_end,
            params.boids_count
    );
    hipDeviceSynchronize();

    move_boids_data_to_cpu(boids);
    swap_buffers();
}

void GPUBoids::reset(const SimulationParameters &params) {
    hipError_t cuda_status = hipMemcpy(m_dev_sim_params, &params, sizeof(boids::SimulationParameters), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    size_t threads_per_block = 1024;
    size_t blocks_num = params.boids_count / threads_per_block + 1;

    ker_reset_simulation<<<blocks_num, threads_per_block>>>(m_dev_sim_params, m_dev_position_old, m_dev_velocity_old, m_dev_orient);
}

void GPUBoids::move_boids_data_to_cpu(Boids &boids) {
    hipError_t cuda_status;
    // TODO: use direct CUDA -> OPENGL
    cuda_status = hipMemcpy(boids.position, m_dev_position, sizeof(glm::vec4) * SimulationParameters::MAX_BOID_COUNT, hipMemcpyDeviceToHost);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(&boids.orientation, m_dev_orient, sizeof(BoidsOrientation), hipMemcpyDeviceToHost);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
}

void GPUBoids::swap_buffers() {
    glm::vec4 *temp_pos = m_dev_position;
    glm::vec3 *temp_vel = m_dev_velocity;

    m_dev_position = m_dev_position_old;
    m_dev_velocity = m_dev_velocity_old;

    m_dev_position_old = temp_pos;
    m_dev_velocity_old = temp_vel;
}