#include "hip/hip_runtime.h"
#include "boids_cuda.h"
#include "hip/hip_runtime.h"

#include <iostream>

// Stores boid's cell id.
__device__ boids::cuda::CellId boid_cell_id[boids::SimulationParameters::MAX_BOID_COUNT];

// Cell id -> cell info id -> count/start in boid_cell_id

// Stores cell info id for each cell id. Cell info id allows querying counts and starts arrays.
__device__ boids::cuda::CellInfoId cell_info_id[boids::SimulationParameters::MAX_CELL_COUNT];

// Stores count of boids inside the boid_cell_id array.
__device__ size_t count[boids::SimulationParameters::MAX_BOID_COUNT];

// Stores starting index of all elements in the queried cell.
__device__ boids::BoidId start[boids::SimulationParameters::MAX_BOID_COUNT];

__device__ boids::cuda::CellId flatten_coords(const boids::SimulationParameters& sim_params, boids::cuda::CellCoord x, boids::cuda::CellCoord y, boids::cuda::CellCoord z) {
    float cell_size = 2 * sim_params.distance;
    boids::cuda::CellCoord grid_size_x = std::ceil(sim_params.aquarium_size.x / cell_size);
    boids::cuda::CellCoord grid_size_y = std::ceil(sim_params.aquarium_size.y / cell_size);

    return x + y * grid_size_x + z * grid_size_x * grid_size_y;
}

void check_cuda_error(const hipError_t &cuda_status, const char *msg) {
    if (cuda_status != hipSuccess) {
        std::cerr << msg << hipGetErrorString(cuda_status) << std::endl;
        std::terminate();
    }
};

boids::cuda::GPUBoids::GPUBoids(const boids::Boids& boids) {
    // Allocate memory on the device using hipMalloc
    size_t array_size = SimulationParameters::MAX_BOID_COUNT * sizeof(glm::vec3);

    hipError_t cuda_status;
    cuda_status = hipMalloc((void**)&m_dev_position, array_size);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_velocity, array_size);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_acceleration, array_size);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_forward, array_size);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_up, array_size);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_right, array_size);
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    cuda_status = hipMemcpy(m_dev_position, boids.position, array_size, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_velocity, boids.velocity, array_size, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_acceleration, boids.acceleration, array_size, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_forward, boids.forward, array_size, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_up, boids.up, array_size, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_right, boids.right, array_size, hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
}

boids::cuda::GPUBoids::~GPUBoids() {
    hipFree(m_dev_position);
    hipFree(m_dev_velocity);
    hipFree(m_dev_forward);
    hipFree(m_dev_forward);
    hipFree(m_dev_up);
    hipFree(m_dev_right);
}
